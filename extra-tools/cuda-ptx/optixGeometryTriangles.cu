#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "CommonStructs.h"


rtDeclareVariable( float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, texcoord,         attribute texcoord, );
rtDeclareVariable( float2, barycentrics,     attribute barycentrics, );

rtBuffer<Core::Vertex> vertex_buffer;
rtBuffer<uint3>   index_buffer;

RT_PROGRAM void triangle_attributes()
{
    const uint3  v_idx = index_buffer[rtGetPrimitiveIndex()];
    const float3 v0    = vertex_buffer[v_idx.x].position;
    const float3 v1    = vertex_buffer[v_idx.y].position;
    const float3 v2    = vertex_buffer[v_idx.z].position;
    const float3 Ng    = optix::cross( v1 - v0, v2 - v0 );

    geometric_normal = optix::normalize( Ng );

    barycentrics = rtGetTriangleBarycentrics();
	// could be used if missing uvs
    //texcoord = make_float3( barycentrics.x, barycentrics.y, 0.0f );

    shading_normal = vertex_buffer[v_idx.y].normal * barycentrics.x + vertex_buffer[v_idx.z].normal * barycentrics.y
        + vertex_buffer[v_idx.x].normal * ( 1.0f-barycentrics.x-barycentrics.y );



    const float2 t0 = vertex_buffer[v_idx.x].uv;
    const float2 t1 = vertex_buffer[v_idx.y].uv;
    const float2 t2 = vertex_buffer[v_idx.z].uv;
    texcoord = make_float3( t1*barycentrics.x + t2*barycentrics.y + t0*(1.0f-barycentrics.x-barycentrics.y) );
    
}

