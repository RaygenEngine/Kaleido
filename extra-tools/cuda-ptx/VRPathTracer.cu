#include "hip/hip_runtime.h"
#include "VRPathTracerAux.h"

#include "commonStructs.h"
#include "random.h"

#include "lambert_brdf.h"
#include "ggx_brdf.h"

#include <optixu/optixu_matrix_namespace.h>

#define FAR 300

struct PerRayData_radiance_VRPT
{
	float3 result;
	float3 radiance;
	float depth_value;
	float3 albedo;
	float3 normal;
	float3 throughput;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int done;
};

// TODO: there is an issue (alignement) with the payload structures (possibly an Optix 6.0 bug?)
struct PerRayData_shadow_VRPT
{
	int in_shadow;
	float3 result;
	float3 radiance;
	float depth_value;
	float3 albedo;
	float3 normal;
	float3 throughput;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int done;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );

rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );


rtBuffer<float4, 2> output_buffer;
rtBuffer<float,  2> depth_buffer;
rtBuffer<float4, 2> input_albedo_buffer;
rtBuffer<float4, 2> input_normal_buffer;


rtDeclareVariable(unsigned int, max_spp, , );
rtDeclareVariable(unsigned int, max_bounces, , );
rtDeclareVariable(unsigned int, sqrt_max_spp, , );

rtDeclareVariable(int, enable_sampling_mask, , );


rtDeclareVariable(float, time_, , );
////////////////////////////////////////// RAY GENERATION PROGRAMS //////////////////////////////////////////

static __device__ __inline__ void ray_gen_eye(const float3& eye, const float3& eye_u, const float3& eye_v, const float3& eye_w, const Matrix3x3& normal_matrix, const uint2& buffer_index)
{
	size_t2 eye_screen = output_buffer.size();
	eye_screen.x = eye_screen.x / 2;

	float2 inv_screen = 1.0f / make_float2(eye_screen) * 2.f;
	float2 pixel = make_float2(launch_index) * inv_screen - 1.f;

	float2 center = make_float2(eye_screen.x / 2.f, eye_screen.y / 2.f);
	float2 point = make_float2(launch_index);
	float distance_from_center = length(point - center);

	uint samples_per_pixel = max_spp;

	if (enable_sampling_mask != 0)
	{

		float2 d = make_float2(0.f);

		float normalized = 1 - (distance_from_center / (eye_screen.x));
		samples_per_pixel = 1 + uint(normalized*(max_spp - 1) / 8);

		auto center_region = eye_screen.x / 2.5;

		if (distance_from_center <= center_region)
		{
			auto dist = (distance_from_center / (center_region));

			normalized = 1 - (dist * dist);

			samples_per_pixel += uint(normalized*(max_spp - 1));
		}

		if (samples_per_pixel == 0)
			return;

	}
	//right eye test
	//if (buffer_index.x > eye_screen.x)
	//{
	//	output_buffer[buffer_index] = make_float4(samples_per_pixel / float(max_spp));
	//	return;
	//}

	float2 jitter_scale = inv_screen / sqrtf(samples_per_pixel);

	float3 result = make_float3(0.0f);
	float3 albedo = make_float3(0.0f);
	float3 normal = make_float3(0.0f);
	float depth_value = 0.f;

	float spp = 0.f;	
	unsigned int seed = tea<16>(eye_screen.x*launch_index.y + launch_index.x, 124144);//fmodf(time_, 1241)
	//if (buffer_index.x > eye_screen.x)
	//{
	//	seed = tea<16>(eye_screen.y*buffer_index.y + buffer_index.x, 124144);//fmodf(time_, 1241)
	//}

	do
	{
		//float2 d = make_float2(0.f);

		//right eye test
		//if(buffer_index.x > eye_screen.x)
		//{
		//	d = pixel;

		//}
		//else
		//{
			unsigned int x = max_spp % sqrt_max_spp;
			unsigned int y = max_spp / sqrt_max_spp;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = pixel + jitter * 0.0025;
		//}

		auto current_ray_origin = eye;
		auto current_ray_direction = normalize(d.x * eye_u + d.y * eye_v + eye_w);

		// Initialze per-ray data
		PerRayData_radiance_VRPT prd;
		prd.result = make_float3(0.f);
		prd.albedo = make_float3(0.f);
		prd.throughput = make_float3(1.f);
		prd.done = false;
		prd.seed = seed;
		prd.depth = 0;

		// Each iteration is a segment of the ray path.  The closest hit will
		// return new segments to be traced here.
		auto max_depth = max_bounces;

		do
		{
			optix::Ray ray = make_Ray(current_ray_origin, current_ray_direction, radiance_ray_type, scene_epsilon, FAR);
			rtTrace(top_object, ray, prd);

			// We have hit the background or a luminaire
			if (prd.done)
			{
				//if(prd.depth < 1)
				//	prd.result = make_float3(1.f);

				break;
			}

			// Russian roulette termination 
			if (prd.depth >= 5)
			{
				float pcont = fmaxf(prd.throughput);
				if (rnd(prd.seed) >= pcont)
					break;
				prd.throughput /= pcont;
			}

			prd.depth++;

			// Update ray data for the next path segment
			current_ray_origin = prd.origin;
			current_ray_direction = prd.direction;

		} while (--max_depth);

		result += prd.result;
		seed = prd.seed;

		albedo += prd.albedo;
		++spp;
		float3 normal_eyespace = (length(prd.normal) > 0.f) ? normalize(normal_matrix * prd.normal) : make_float3(0., 0., 1.);
		normal += normal_eyespace;

		depth_value = prd.depth_value;

	} while (--samples_per_pixel);
	result = result / spp;
	output_buffer[buffer_index]       = make_float4(clamp(result, 0.f, 1.f), 1.0f);
	input_albedo_buffer[buffer_index] = make_float4(albedo / spp, 1.0f);
	input_normal_buffer[buffer_index] = make_float4(normal / spp, 1.0f);

	depth_buffer[buffer_index] = depth_value / FAR;// log2(max(1e-6, 1.0 + depth_value)) * (2.0 / log2(FAR + 1.0)) - 1.0;
}


rtDeclareVariable(float3, eye_L, , );
rtDeclareVariable(float3, U_L, , );
rtDeclareVariable(float3, V_L, , );
rtDeclareVariable(float3, W_L, , );
rtDeclareVariable(Matrix3x3, normal_matrix_L, , );

// different eye raygens for future path reprojection work 
RT_PROGRAM void pathtrace_vr_left_eye()
{
	uint2 buffer_index = make_uint2(launch_index.x, launch_index.y);

	ray_gen_eye(eye_L, U_L, V_L, W_L, normal_matrix_L, buffer_index);
}

rtDeclareVariable(float3, eye_R, , );
rtDeclareVariable(float3, U_R, , );
rtDeclareVariable(float3, V_R, , );
rtDeclareVariable(float3, W_R, , );
rtDeclareVariable(Matrix3x3, normal_matrix_R, , );

// to access results from left eye use the left side of the output/albedo/normal buffer
RT_PROGRAM void pathtrace_vr_right_eye()
{
	uint2 buffer_index = make_uint2(launch_index.x + (output_buffer.size().x/2), launch_index.y);

	//output_buffer[buffer_index] = output_buffer[launch_index];
	//input_albedo_buffer[buffer_index] = input_albedo_buffer[launch_index];
	//input_normal_buffer[buffer_index] = input_normal_buffer[launch_index];

	ray_gen_eye(eye_R, U_R, V_R, W_R, normal_matrix_R, buffer_index);
}

////////////////////////////////////////// RAY GENERATION PROGRAMS //////////////////////////////////////////


////////////////////////////////////////// CLOSEST HIT DIFFUSE PROGRAM //////////////////////////////////////////

// Bindless mat texture IDs.
rtDeclareVariable(int, Albedo_mapId, , );
rtDeclareVariable(int, Emission_mapId, , );
rtDeclareVariable(int, SpecularParameters_mapId, , );
rtDeclareVariable(int, Bump_mapId, , );
rtDeclareVariable(int, Noise_mapId, , );

rtDeclareVariable(int, hasNormalMap, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(PerRayData_radiance_VRPT, current_prd, rtPayload, );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(Core::BasicLight, light, ,);

rtDeclareVariable(float, forced_metal, , );
rtDeclareVariable(float, forced_roughness, , );
rtDeclareVariable(float, forced_reflectance, , );
rtDeclareVariable(float, light_intensity, , );

RT_PROGRAM void surface_shading()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	float3 hit_point = ray.origin + t_hit * ray.direction;

	// shading normal
	float3 ns = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	optix::Onb onb_local(ns);
	if (hasNormalMap != -1)
	{
		//float3 sample = make_float3(rtTex2D<float4>(Bump_mapId, texcoord.x, texcoord.y)) * 2 - 1;
		//onb_local.inverse_transform(sample);
		//ns = normalize(sample);
	}

	const float3 albedo = make_float3(rtTex2D<float4>(Albedo_mapId, texcoord.x, texcoord.y));
	// x : refle, y: roughness, z: metal
	const float4 specular_parameters = rtTex2D<float4>(SpecularParameters_mapId, texcoord.x, texcoord.y);

	if (current_prd.depth == 0)
	{
		current_prd.depth_value = abs(t_hit);

		current_prd.albedo = albedo;
		current_prd.normal = ns;
	}

	float reflectivity = forced_reflectance;//specular_parameters.x;
	float roughness = forced_roughness;// force  specular_parameters.y;
	float metallic = forced_metal;// specular_parameters.z;



	float3 diffuse_color = (1.0 - metallic) * albedo;
	float3 f0 = make_float3(0.16) * reflectivity * reflectivity * (1.0 - metallic) + (albedo * metallic);


	// incoming (surface space)
	float3 wi = world_to_surface(onb_local, -ray.direction);

	// DIRECT ILLUMINATION

	 //Calculate properties of light sample 
	const float  Ldist = length(light.pos - hit_point);
	const float3 L = normalize(light.pos - hit_point);
	const float  nDl = dot(ns, L);

	float3 luminance = make_float3(0.0f);

	//cast shadow ray
	if (nDl > 0.0f)
	{
		PerRayData_shadow_VRPT shadow_prd;
		shadow_prd.in_shadow = false;

		Ray shadow_ray = make_Ray(hit_point, L, shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
		rtTrace(top_shadower, shadow_ray, shadow_prd);

		if (!shadow_prd.in_shadow)
			luminance = light.color * light_intensity * nDl / (Ldist*Ldist);
	}

	// direct diffuse and specular
	float3 wl = world_to_surface(onb_local, L);

	float3 wh_d = normalize(wi + wl);

	float3 F_d = Fresnel_Schlick(max(0.0, dot(wi, wh_d)), f0);

	float3 fr_d = MicrofacetGGX_Evaluate(roughness, F_d, wh_d, wi, wl);

	float3 fd_d = diffuse_color / M_PIf;

	current_prd.result = current_prd.result + (luminance * current_prd.throughput *(fd_d + fr_d)); //lerp(fd_d, fr_d, (F_d.x+F_d.y+F_d.z)/3.f));


	// INDIRECT ILLUMINATION



	// reflect or transmit
	float3 F_p = Fresnel_Schlick(fabs(wi.y), f0);

	float2 sample = make_float2(rnd(current_prd.seed), rnd(current_prd.seed));
	// halfway (ss)
	float3 wh;
	// outgoing (ss)
	float3 wo;
	// pdf of outgoing 
	float pdf;
	MicrofacetDistribution_GGX_SampleHalfwayVector(wi, roughness, sample, wh, wo, pdf);

	float3 F_i = Fresnel_Schlick(max(0.0, dot(wi, wh)), f0);
	float k_s_i = (F_i.x + F_i.y + F_i.z) / 3.f;

	if(rnd(current_prd.seed) > k_s_i)
	{
		float z1 = rnd(current_prd.seed);
		float z2 = rnd(current_prd.seed);
		float3 p;
		cosine_sample_hemisphere(z1, z2, p);
		onb_local.inverse_transform(p);
		current_prd.direction = p;

		current_prd.throughput *= diffuse_color;
	}
	else
	{

		if (roughness <= 0.01)
		{
			float3 wo = optix::reflect(-wi, make_float3(0,1,0));

			// next path segment
			current_prd.origin = hit_point;
			current_prd.direction = surface_to_world(onb_local, wo);

			return;
		}

		//float2 sample = make_float2(rnd(current_prd.seed), rnd(current_prd.seed));
		//// halfway (ss)
		//float3 wh;
		//// outgoing (ss)
		//float3 wo;
		//// pdf of outgoing 
		//float pdf;
		//MicrofacetDistribution_GGX_SampleHalfwayVector(wi, roughness, sample, wh, wo, pdf);

		//float3 F_i = Fresnel_Schlick(max(0.0, dot(wi, wh)), f0);
		//float k_s_i = (F_i.x + F_i.y + F_i.z) / 3.f;

		float3 fr_i = MicrofacetGGX_Evaluate(roughness, F_i, wh, wi, wo) / pdf;

		current_prd.throughput *= fr_i*fabs(wo.y);
		//current_prd.throughput *= fabs(dot(ns, current_prd.direction));

		current_prd.direction = surface_to_world(onb_local, wo);
	}
	//// next path segment
	current_prd.origin = hit_point;
}

////////////////////////////////////////// CLOSEST HIT DIFFUSE PROGRAM //////////////////////////////////////////


////////////////////////////////////////// MISS PROGRAM //////////////////////////////////////////

rtDeclareVariable(float3, bg_color, , );

// Bindless sky texture id.
rtDeclareVariable(int, sky_mapId, , );

RT_PROGRAM void miss()
{
	//current_prd.radiance = optix::make_float3(optix::rtTexCubemap<float4>(sky_mapId, ray.direction.x, ray.direction.y, ray.direction.z));
	//current_prd.radiance = bg_color;

	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));

	current_prd.result = current_prd.result + (current_prd.throughput * make_float3(rtTex2D<float4>(sky_mapId, u, v)));
	current_prd.done = true;

	// TODO: Find out what the albedo buffer should really have. For now just set to black for misses.
	if (current_prd.depth == 0)
	{
		current_prd.depth_value = FAR;

		current_prd.albedo = make_float3(0, 0, 0);
		current_prd.normal = make_float3(0, 0, 0);
	}
}

////////////////////////////////////////// MISS PROGRAM //////////////////////////////////////////


////////////////////////////////////////// EXCEPTION PROGRAM //////////////////////////////////////////

rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.f);
}

////////////////////////////////////////// EXCEPTION PROGRAM //////////////////////////////////////////


////////////////////////////////////////// ANY HIT SHADOW PROGRAM //////////////////////////////////////////

rtDeclareVariable(PerRayData_shadow_VRPT, current_prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	current_prd_shadow.in_shadow = 1;
	rtTerminateRay();
}

////////////////////////////////////////// ANY HIT SHADOW PROGRAM //////////////////////////////////////////