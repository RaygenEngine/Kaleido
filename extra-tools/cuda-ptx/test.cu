#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "test.h"

using namespace optix;

////////////////////////////////////////// RAY GENERATION PROGRAM //////////////////////////////////////////

rtDeclareVariable(PerRayData_result, prd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, result_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;

	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);
	optix::Ray ray(ray_origin, ray_direction, result_ray_type, scene_epsilon);
	PerRayData_result prd;

	rtTrace(top_object, ray, prd);
	output_buffer[launch_index] = make_color(prd.result);
}

////////////////////////////////////////// MISS PROGRAM //////////////////////////////////////////

rtDeclareVariable(float3, bg_color, , );
// Bindless sky texture id.
rtDeclareVariable(int, sky_mapId, , );

RT_PROGRAM void miss()
{
	//prd_radiance.result = optix::make_float3(optix::rtTexCubemap<float4>(sky_mapId, ray.direction.x, ray.direction.y, ray.direction.z));
	//prd_radiance.result = bg_color;

	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	prd.result = make_float3(rtTex2D<float4>(sky_mapId, u, v));
}

////////////////////////////////////////// EXCEPTION PROGRAM //////////////////////////////////////////

rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}

////////////////////////////////////////// CLOSEST HIT PROGRAM //////////////////////////////////////////

// from intersection
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

// Bindless mat texture IDs.
rtDeclareVariable(int, Albedo_mapId, , ); 
rtDeclareVariable(int, Emission_mapId, , );
rtDeclareVariable(int, SpecularParameters_mapId, , );
rtDeclareVariable(int, Bump_mapId, , );
rtDeclareVariable(int, Noise_mapId, , );

rtDeclareVariable(int, hasNormalMap, , );

rtDeclareVariable(int, mode, , );

// very basic unoptimized surface test shader
RT_PROGRAM void closest_hit()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	const float4 albedo = rtTex2D<float4>(Albedo_mapId, texcoord.x, texcoord.y);
	const float4 emission = rtTex2D<float4>(Emission_mapId, texcoord.x, texcoord.y);
	const float4 specular_parameters = rtTex2D<float4>(SpecularParameters_mapId, texcoord.x, texcoord.y);
	const float4 bump = rtTex2D<float4>(Bump_mapId, texcoord.x, texcoord.y);
	const float noise = rtTex2D<float>(Noise_mapId, texcoord.x, texcoord.y);
	
	switch (mode)
	{
		case 0: // albedo
			prd.result = make_float3(albedo);
			break;

		case 1: // emission
			prd.result = make_float3(emission);
			break;

		case 2: // reflectivity
			prd.result = make_float3(specular_parameters.x, specular_parameters.x, specular_parameters.x);
			break;

		case 3: // roughness
			prd.result = make_float3(specular_parameters.y, specular_parameters.y, specular_parameters.y);
			break;

		case 4: // metallic
			prd.result = make_float3(specular_parameters.z, specular_parameters.z, specular_parameters.z);
			break;

		case 5: // world normal
			prd.result = ffnormal;
			break;

		case 6: // normal map
			prd.result = make_float3(bump.x, bump.y, bump.z);
			break;

		case 7: // final normal
			if (hasNormalMap)
			{
				optix::Onb onb(ffnormal);
				float3 sample = make_float3(bump.x, bump.y, bump.z) * 2 - 1;
				onb.inverse_transform(sample);
				ffnormal = normalize(sample);
			}
			prd.result = ffnormal;
			break;

		case 8: // uv
			prd.result = make_float3(texcoord.x, texcoord.y, 0);
			break;

		case 9: // height
			prd.result = make_float3(bump.w, bump.w, bump.w);
			break;

		case 10: // translucency
			prd.result = make_float3(specular_parameters.w, specular_parameters.w, specular_parameters.w);
			break;

		case 11: // ambient occlusion
			prd.result = make_float3(emission.w, emission.w, emission.w);
			break;

		case 12: // opacity
			prd.result = make_float3(albedo.w, albedo.w, albedo.w);
			break;

		case 13: // noise
			prd.result = make_float3(noise, noise, noise);
			break;
	}
}

////////////////////////////////////////// ANY HIT PROGRAM //////////////////////////////////////////

RT_PROGRAM void any_hit()
{

}