#include "hip/hip_runtime.h"
#include "tutorial.h"


rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );

rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable(rtObject, top_object, , );

//
// Pinhole camera implementation
//

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;

	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);
	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);
	output_buffer[launch_index] = make_color(prd.result);
} 

//
// Returns solid color for miss rays
//

rtDeclareVariable(float3, bg_color, , );

// Bindless sky texture id.
rtDeclareVariable(int, sky_mapId, , );

RT_PROGRAM void miss()
{
	//prd_radiance.result = optix::make_float3(optix::rtTexCubemap<float4>(sky_mapId, ray.direction.x, ray.direction.y, ray.direction.z));
	//prd_radiance.result = bg_color;

	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	prd_radiance.result = make_float3(rtTex2D<float4>(sky_mapId, u, v));
} 

//
// Set pixel to solid color upon failur
//

rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
